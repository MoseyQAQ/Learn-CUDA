/*
Hello Word in CUDA using multiple threads
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU()
{   
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    printf("Hello World from GPU! Block ID: %d, Thread ID: %d\n", bid, tid);
}

int main() {
    helloFromGPU<<<4, 4>>>();
    hipDeviceSynchronize();
    return 0;
}