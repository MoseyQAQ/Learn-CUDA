/*
Hello Word in CUDA using multiple threads and multiple dimension
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU()
{   
    const int bid = blockIdx.x;
    const int tidx = threadIdx.x;
    const int tidy = threadIdx.y;
    printf("Hello World from GPU! Block ID: %d, Thread ID: (%d, %d)\n", bid, tidx, tidy);
}

int main() {
    const dim3 block_size(4, 2); // 4, 2, 1
    helloFromGPU<<<1, block_size>>>();
    hipDeviceSynchronize();
    return 0;
}