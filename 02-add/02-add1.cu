
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 4.56;
const double c = 5.79;

void __global__ add(const double *x, const double *y, double *z);
void check(const double *z, const int N);

int main(void){
    const int N = 100000000;
    const int M = sizeof(double) * N;
    double *cpu_x = (double*) malloc(M);
    double *cpu_y = (double*) malloc(M);
    double *cpu_z = (double*) malloc(M);

    for (int i = 0; i < N; i++) {
        cpu_x[i] = a;
        cpu_y[i] = b;
    }

    double *gpu_x, *gpu_y, *gpu_z;
    hipMalloc((void**)&gpu_x, M);
    hipMalloc((void**)&gpu_y, M);
    hipMalloc((void**)&gpu_z, M);
    // cudaMemcpy(gpu_x, cpu_x, M, cudaMemcpyHostToDevice);
    hipMemcpy(gpu_x, cpu_x, M, hipMemcpyDeviceToHost); // error
    hipMemcpy(gpu_y, cpu_y, M, hipMemcpyHostToDevice);

    const int block_size = 128;
    const int grid_size = N / block_size;
    add<<<grid_size, block_size>>>(gpu_x, gpu_y, gpu_z);
    hipMemcpy(cpu_z, gpu_z, M, hipMemcpyDeviceToHost);
    check(cpu_z, N);

    free(cpu_x);
    free(cpu_y);
    free(cpu_z);
    hipFree(gpu_x);
    hipFree(gpu_y);
    hipFree(gpu_z);
    return 0;
}

void __global__ add(const double *x, const double *y, double *z){
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    z[n] = x[n] + y[n];
}

void check(const double *z, const int N) {
    bool has_error = false;
    for (int i = 0; i < N; i++) {
        if (fabs(z[i] - c) > EPSILON) {
            has_error = true;
        }
    }
    if (!has_error) {
        printf("check ok\n");
    }
    else {
        printf("check error\n");
    }
}